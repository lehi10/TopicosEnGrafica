#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>

//#include <cv.h>
//#include <highgui.h>


using namespace cv;
using namespace std;

string originalImage ="img6.jpg";
string faceFilter = "face6.jpg";

float maxGlob =0;
float minGlob =0;




#define CHECK(call){const hipError_t error = call;if (error != hipSuccess){printf("Error: %s:%d, ", __FILE__, __LINE__);printf("code:%d, reason: %s\n", error, hipGetErrorString(error));exit(1);}}


template <class T>
T* cuda_array(int n){
  T* arr;
  size_t tam = n * sizeof(T);
  CHECK(hipMalloc(&arr, tam));
  return arr;
}

template <class T>
void cuda_H2D(T* h_arr, T* d_arr, int n){
  size_t tam = n * sizeof(T);
  CHECK(hipMemcpy(d_arr, h_arr, tam, hipMemcpyHostToDevice));
}


template <class T>
void cuda_D2H(T* d_arr, T* h_arr, int n){
  size_t tam = n * sizeof(T);
  CHECK(hipMemcpy(h_arr, d_arr, tam, hipMemcpyDeviceToHost));
}


__global__ void conv(unsigned char* d_data, float* d_kernel, int k, int p, int pos_k, int pos_p, int h, int w){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if(i < w && j < h){
    float sum_c1 = 0;
    float sum_c2 = 0;
    float sum_c3 = 0;
    for (size_t r = 0; r < k; r++) {
      for (size_t c = 0; c < p; c++) {
      
        sum += d_data[(j - pos_k + r) * w + (i - pos_p + c)] * d_kernel[r * p + c];

        
      }
    }
    d_data[j * w + i ] = sum;
  }
}


Mat aplicar_filtro(Mat& img_i, float* kernel, int k , int p, int i, int j){
  Mat img = img_i.clone();
  int h, w;
  h = img.size().height;
  w = img.size().width;


  float block_size = 16;
  dim3 block =  dim3(block_size, block_size, 1);
  dim3 grid =  dim3(ceil(w / block_size), ceil(h / block_size), 1);

  unsigned char* data = (unsigned char*)img.data;
  unsigned char* d_data = cuda_array<unsigned char>(h * w );
  float* d_kernel = cuda_array<float>(k * p);


  cuda_H2D(data, d_data, h * w );
  cuda_H2D(kernel, d_kernel, k * p);
  
  conv<<<grid, block>>>(d_data, d_kernel, k, p, i, j, h, w);
  
  hipDeviceSynchronize();
  cuda_D2H(d_data, data, h * w );
  hipFree(d_kernel);
  return img;

}



float normalize(float input,float minV, float maxV)
{    

    return (input - minV) / (maxV - minV);
}

void templateMatchingInGray()
{
	cout<<"Using Gray Scale : "<<endl;
	Mat image, gray_image, 		//Original Image
		imgFilter,imgFilterGray;//Filter Image

	// Original Image 
	image = imread( originalImage, 1 );
	resize(image, image,cv::Size(), 0.6, 0.6);
	
	cvtColor( image, gray_image, CV_BGR2GRAY );
	//GaussianBlur(gray_image,gray_image,Size(5,5),10.0); 
	//Canny( gray_image, gray_image, 50, 300, 3);

		
	//Filter Image
	imgFilter = imread( faceFilter, 1 );
	resize(imgFilter, imgFilter,cv::Size(), 0.6, 0.6);
	
	cvtColor( imgFilter, imgFilterGray, CV_BGR2GRAY );
	GaussianBlur(imgFilterGray,imgFilterGray,Size(7,7),150.0); 
	//Canny( imgFilterGray,imgFilterGray, 50, 300, 3);
	
	
	int wFilter =imgFilterGray.cols;
	int hFilter =imgFilterGray.rows;	
	
	int w=image.cols;
	int h=image.rows;

	int wMiddlePointFilter=wFilter/2;
	int hMiddlePointFilter=hFilter/2;
	
	
	//Template matching Matrix
	Mat imgTM(h,w, CV_8UC1, Scalar(0));
	
	int filterScale=sqrt(wFilter*hFilter*255*255);

	cout<<"Buscando en la imagen"<<endl;
	for(int i=0+hMiddlePointFilter;i<(h-hMiddlePointFilter);i+=2)
	{		
		for(int j=0+wMiddlePointFilter;j<w-wMiddlePointFilter;j+=2)
		{
			int factor=0;
			for(int x=0  ;x<hFilter;x++)
			{
				for(int y=0;y<wFilter;y++)
				{
					int pixelFilter	=imgFilterGray.at<uchar>(x, y);
					
					int _x = (i-hMiddlePointFilter)+x;
					int _y = (j-wMiddlePointFilter)+y;
					
					int pixel =gray_image.at<uchar>(_x, _y);
					
					factor+=pow(pixel-pixelFilter,2);
				}
			}
			float distance = normalize(sqrt(factor),0,filterScale);
			imgTM.at<uchar>(i,j) = 255-distance*255;
			if(distance < 0.15 )
			{	
				//cout<<distance<<endl;
				Point point_1 = Point(j+wMiddlePointFilter,i+hMiddlePointFilter);
				Point point_2 = Point(j-wMiddlePointFilter,i-hMiddlePointFilter);
				
				rectangle(image,point_1,point_2,Scalar(0,0,255),1);
				
			}
		}
	}
	
	imwrite( "result.jpg", image);
	
	//namedWindow( "TM", CV_WINDOW_AUTOSIZE/2);			
	//namedWindow( "Original", CV_WINDOW_AUTOSIZE/2);
	

	//imshow( "TM", imgTM );
	//imshow( "Original", image );	
	
	//waitKey(0);
}


void templateMatchingInColor()
{
	cout<<"Using Colors in RGB"<<endl;
	Mat image, gray_image, 		//Original Image
		imgFilter,imgFilterGray;//Filter Image

	
	// Original Image 
	image = imread( originalImage, 1 );
	resize(image, image,cv::Size(), 0.6, 0.6);


	//Filter Image
	imgFilter = imread( faceFilter, 1 );
	resize(imgFilter, imgFilter,cv::Size(), 0.6, 0.6);
	
	cvtColor(imgFilter, imgFilter,COLOR_BGR2RGB);
	GaussianBlur(imgFilter,imgFilter,Size(7,7),150.0); 
	
	int wFilter =imgFilter.cols;
	int hFilter =imgFilter.rows;	

	int w=image.cols;
	int h=image.rows;

	int wMiddlePointFilter=wFilter/2;
	int hMiddlePointFilter=hFilter/2;
	
	//Template matching Matrix
	Mat imgTM(h,w, CV_8UC1, Scalar(0));
	
	int filterScale=sqrt(wFilter*hFilter*255*255);
	
	cout<<"Buscando en la imagen"<<endl;
	for(int i=0+hMiddlePointFilter;i<(h-hMiddlePointFilter);i+=2)
	{		

		for(int j=0+wMiddlePointFilter;j<w-wMiddlePointFilter;j+=2)
		{
			float factor=0;
			for(int x=0  ;x<hFilter;x++)
			{
				for(int y=0;y<wFilter;y++)
				{
					
					//Punto en el Filtro
					Vec3b pixelA =imgFilter.at<Vec3b>(x,y);
					//Punto en la imagen
					int _x = (i-hMiddlePointFilter)+x;
					int _y = (j-wMiddlePointFilter)+y;
					Vec3b pixelB =image.at<Vec3b>(_x, _y);
									
					//int pixVal_1 = normalize(int(pixelA[0]) *int(pixelA[1])*int(pixelA[2]),0,16581375)*255;
					//int pixVal_2 = normalize(int(pixelB[0]) *int(pixelB[1])*int(pixelB[2]),0,16581375)*255;
					int pixVal_1 = normalize(int(pixelA[0]) +int(pixelA[1])+int(pixelA[2]),0,765)*255;
					int pixVal_2 = normalize(int(pixelB[0]) +int(pixelB[1])+int(pixelB[2]),0,765)*255;
					
					factor+=pow(pixVal_1-pixVal_2,2);
					
					

					
				}
			}
			float distance = normalize(sqrt(factor),0,filterScale);
			imgTM.at<uchar>(i,j) = distance*255;			
			
			
			if(distance < 0.14 )
			{	
				
				Point point_1 = Point(j+wMiddlePointFilter,i+hMiddlePointFilter);
				Point point_2 = Point(j-wMiddlePointFilter,i-hMiddlePointFilter);
				rectangle(image,point_1,point_2,Scalar(0,0,255),1);
				
			}
		}
	}

	imwrite( "result.jpg", image);

	//namedWindow( "TM", CV_WINDOW_AUTOSIZE/2);			
	//namedWindow( "Original", CV_WINDOW_AUTOSIZE/2);

	//imshow( "TM", imgTM );
	//imshow( "Original", image );	
	
	
	//waitKey(0);
}

int main( int argc, char** argv )
{	
	templateMatchingInColor();
	//templateMatchingInGray();

	return 0;
}




