#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include "CImg.h"

using namespace std;
using namespace cimg_library;

#include <iostream>
#define BLOCK_SIZE 32
#define WA 512   
#define HA 512     
#define HC 3     
#define WC 3
#define WB (WA - WC + 1)
#define HB (HA - HC + 1)
// -lX11
using namespace std;
__global__ void Convolucion(float* A, float* B, float* C, int numARows, int numACols, int numBRows, int numBCols, int numCRows, int numCCols)
{
	int col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
	int row = blockIdx.y * (BLOCK_SIZE - WC + 1) + threadIdx.y;
	int row_i = row - WC + 1;
	int col_i = col - WC + 1;

	float tmp = 0;

	__shared__ float shm[BLOCK_SIZE][BLOCK_SIZE];

	if (row_i < WA && row_i >= 0 && col_i < WA && col_i >= 0)
	{
		shm[threadIdx.y][threadIdx.x] = A[col_i * WA + row_i];
	}
	else
	{
		shm[threadIdx.y][threadIdx.x] = 0;
	}

	__syncthreads();

	if (threadIdx.y < (BLOCK_SIZE - WC + 1) && threadIdx.x < (BLOCK_SIZE - WC + 1) && row < (WB - WC + 1) && col < (WB - WC + 1))
	{
		for (int i = 0; i< WC;i++)
			for (int j = 0;j<WC;j++)
				tmp += shm[threadIdx.y + i][threadIdx.x + j] * C[j*WC + i];
		B[col*WB + row] = tmp;
	}
}


int main(int argc, char** argv)
{
	CImg<unsigned char> src("a.png");
    int width = src.width();
    int height = src.height();
    
    unsigned long size = src.size();
    
    int A[width][width];
    
	  
	
	srand(2006);
	hipError_t error;
	hipEvent_t start_G, stop_G;

	hipEventCreate(&start_G);
	hipEventCreate(&stop_G);

	unsigned int size_A = WA * HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A = (float*)malloc(mem_size_A);

	unsigned int size_B = WB * HB;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float* h_B = (float*)malloc(mem_size_B);

	unsigned int size_C = WC * HC;
	unsigned int mem_size_C = sizeof(float) * size_C;
	float* h_C = (float*)malloc(mem_size_C);

	//Sobel derivada en x
	/*h_C[0]=-1;h_C[1]=0;h_C[2]=1;
	h_C[3]=-2;h_C[4]=0;h_C[5]=2;
	h_C[6]=-1;h_C[7]=0;h_C[8]=1;*/
	
	//Sobel derivada en y
	/*h_C[0]=-1;h_C[1]=-2;h_C[2]=-1;
	h_C[3]=0;h_C[4]=0;h_C[5]=0;
	h_C[6]=1;h_C[7]=2;h_C[8]=1;*/
	
	//filtro de prewitt
	h_C[0]=-1;h_C[1]=0;h_C[2]=1;
	h_C[3]=-1;h_C[4]=0;h_C[5]=1;
	h_C[6]=-1;h_C[7]=0;h_C[8]=1;
	

	//int B[xF][yF]={{0,0,0},{0,1,0},{0,0,0}};
	int n=0;
	for (int x=0;x<width;x++){
		for (int y=0;y<width;y++){
			unsigned char r =(float)src(x,y,0,0);
			unsigned char g =(float)src(x,y,0,1);
			unsigned char b =(float)src(x,y,0,2);
									
			int _gray = (unsigned int)((float)(r + g + b) / 3.0f + 0.5);
			
			src(x,y,0,0)=_gray;
			src(x,y,0,1)=_gray;
			src(x,y,0,2)=_gray;
			
			A[x][y] =_gray;
			h_A[n]=_gray;
			cout<<_gray<<" ";
			n++;
		}	
		cout<<endl;
	}
	
	src.display();
	float* d_A;
	float* d_B;
	float* d_C;

	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_B, mem_size_B);
	hipMalloc((void**)&d_C, mem_size_C);

	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, mem_size_C, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((WB - 1) / (BLOCK_SIZE - WC + 1), (WB - 1) / (BLOCK_SIZE - WC + 1));

	Convolucion << < grid, threads >> >(d_A, d_B, d_C, HA, WA, HB, WB, HC, WC);
	
	hipMemcpy(h_B, d_B, mem_size_B, hipMemcpyDeviceToHost);

	for (int i = 0;i < HB;i++)
	{
		for (int j = 0;j < WB;j++)
		{
			src(i,j,0,0)=h_B[i*HB + j];
			src(i,j,0,1)=h_B[i*HB + j];
			src(i,j,0,2)=h_B[i*HB + j];
			
			cout<<h_B[i*HB + j]<<" ";
//			printf("%f ", h_B[i*HB + j]);
		}
		printf("\n");
	}
	

	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	src.display();
	return EXIT_SUCCESS;
}
